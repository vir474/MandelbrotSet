#include "hip/hip_runtime.h"
// 
//	kernel.cu - Kernels for fractal (Julia and Mandelbrot) set generation
//

#include <hip/hip_runtime.h>
#include <>
#include <GL/freeglut.h>
#include <stdio.h>
#include <stdlib.h> // including: #define EXIT_SUCCESS    0  #define EXIT_FAILURE    1

struct hipComplex 
{
    GLdouble   r;
    GLdouble   i;
    __device__ hipComplex( GLdouble a, GLdouble b ) : r(a) , i( b)  { }
    __device__ GLdouble magnitude2( void ) 
    {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) 
    {
        return hipComplex( r*a.r - i*a.i, i*a.r + r*a.i) ;
    }
    __device__ hipComplex operator+(const hipComplex& a) 
    {
        return hipComplex( r+a.r, i+a.i) ;
    }
};

__device__ float toColor (int i)
{
	float intensity [10] = {0, 0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f, 0.9f};

	return intensity[i%10];
}

//=================================================================================================
//
//	Mandelbrot and Julia set GPU device functions
//
//=================================================================================================

__device__ int mandelbrot(int row, int col, int width, int height, GLdouble Rmin, GLdouble Rmax, GLdouble Imin, GLdouble Imax, int nIterations)
{
	// Normalize (row, col) to {(R,I) | Rmin < R < Rmax, Imin < I < Imax }
	GLdouble R = ((Rmax - Rmin)/(float)width) * (float)col + Rmin;
    //float I = ((Imax - Imin)/(float)height) * (float)row + Imin;
	GLdouble I = Imax - ((Imax - Imin)/(float)height) * (float)row;
	//if (I < 0.00001f && I > -0.00001f) I = 0.0f;

    hipComplex c(R, I);
    hipComplex a(R, I) ;
    int i = 0;
    for (i = 0; i < nIterations; i++) 
    {
        a = a * a + c;
        if (a. magnitude2() > 4.0)
            return i;
    }

    return i;
}

/////////////////////////////////////////////////////////////////////////////////
//
//	Kernel : kernel1
//
//		Computes colors of width by height pixels representing Julia set in
//		{(u,v) | -scale < u < scale, -scale < v < scale }
//
//	Thread grid requirments:
//
//		1) 2D grid of 2D thread blocks covering width by height pixels
//		2) one pixel per thread computing
//
/////////////////////////////////////////////////////////////////////////////////

__global__ void Mandelbrot_kernel(int *ptr,int width, int height, GLdouble Rmin, GLdouble Rmax, GLdouble Imin, GLdouble Imax, int nIterations) 
{
    // map from threadIdx/BlockIdx to pixel position
    int col = threadIdx.x + blockIdx.x * blockDim.x;	// column index to the width X height pixels
    int row = threadIdx.y + blockIdx.y * blockDim.y;	// row index to the width X height pixels

    // Assuming the origin of the width X height pixels is at upper-left corner
    if (row < height && col < width) 
    {
		// Calculate Mandelbrot value at (x,hy) position
		int index = (col + (height-row-1) * width);
		ptr[index] = mandelbrot(row, col, width, height, Rmin, Rmax, Imin, Imax, nIterations);
/*
		if (*ptr[index] == nIterations)
                {}
            else {                 
                       if(cnt>=0&&cnt<=31)   {b=cnt*4; g=cnt*8; r=0;  }
                  else if(cnt>=32&&cnt<=63)  {b=200; g=500-cnt*8; r=0;  }                      
		  else if(cnt>=64&&cnt<=95)  {b=200; g=0; r=(cnt-64)*4;}
		  else if(cnt>=96&&cnt<=127) {r=200; g=0; b=1000-cnt*8;}
		  else if(cnt>=128&&cnt<=159){r=200; g=(cnt-128)*8; b=0;}
		  else if(cnt>=160&&cnt<=191){g=200; r=1500-cnt*8; b=0;}
		  else if(cnt>=192&&cnt<=223){g=200; r=0; b=(cnt-192)*8;}
          else if(cnt>=224&&cnt<=255){g=230; r=(cnt-224)*8; b=256;}


                 //to change color by prssing key 'c'
                   	
                                tr=r;tb=b;tg=g;
                                switch(c)
				{
				case 0: break;
				case 1: r=tb;b=tr;break;
				case 2: r=tg;g=tr;break;
				case 3: b=tg;g=tb;break;
				case 4: r=tg; g=tb; b=tr; break;
				case 5: r=tb; g=tr; b=tg; break;
                      
				}	
			
			   
                    
                   glColor3f(r/256,g/256,b/256);
                   
                glVertex3d(col - nx / 2, row - ny / 2, 0.0f);
*/
	}
}


 #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//=======================================================================================================
//
//	Compute Julia set using CUDA
//
//=======================================================================================================

#define block_size (16)

//=======================================================================================================
//
//	Compute Mandelbrot set using CUDA
//
//=======================================================================================================

extern "C" int cuComputeMandelbrotSet (int *ptr,int width, int height, GLdouble Rmin, GLdouble Rmax, GLdouble Imin, GLdouble Imax, int nIterations)
{
	
	printf("@cuComputeMandelbrotSet %d == %d\n",width, height);
	int *d_ptr = 0;
    
	hipError_t cudaStatus;

	hipDeviceReset();

	// Make sure CUDA device 0 is available
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }
 
    // Allocate device memory
    if (hipMalloc((void **)&d_ptr, width * height * sizeof(int)) != hipSuccess)
    {
		printf("cuda mem failed");
		
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
		return EXIT_FAILURE;
    }
   
	hipDeviceSynchronize();
	// Copy host memory to device
	cudaStatus = hipMemcpy (d_ptr, ptr, width*height*sizeof(int), hipMemcpyHostToDevice);
	 if (cudaStatus != hipSuccess)
    {
        printf("hipMemcpy (d_ptr, ptr) returned error code\n", cudaStatus);
        exit(EXIT_FAILURE);
    }

	// Setup execution parameters and call kernel
    dim3 block(block_size, block_size);
    dim3 grid ((width+block_size-1)/block_size, (height+ block_size-1)/block_size);
	Mandelbrot_kernel<<< grid, block >>>(d_ptr, width, height, Rmin, Rmax, Imin, Imax, nIterations); 
	//gpuErrchk( hipPeekAtLastError() ); 
	//gpuErrchk( hipDeviceSynchronize() );

	// Copy result from device to host
    cudaStatus = hipMemcpy(ptr, d_ptr, width*height*sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        printf("hipMemcpy (ptr, d_ptr) returned error code %d\n", cudaStatus);
        exit(EXIT_FAILURE);
    }

	
    // Device memory clean up
    if (hipFree(d_ptr) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (d_ptr)\n");
		return EXIT_FAILURE;
    }
	return 0;
}
/*
int MandelbrotSet_GPU_GL (float *d_dst, int width, int height, float Rmin, float Rmax, float Imin, float Imax, int nIterations)
{
	// Setup execution parameters and call kernel
    dim3 block(block_size, block_size);
    dim3 grid ((width+block_size-1)/block_size, (height+ block_size-1)/block_size);
	Mandelbrot_kernel<<< grid, block >>>(d_dst, width, height, Rmin, Rmax, Imin, Imax, nIterations); 

	return 0;
}
*/

extern "C" bool resetCUDADevice()
{
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return false;
    }

	return true;
}